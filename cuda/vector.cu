#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define NUM 100

__global__ void add (int *a, int *b, int *c)
{
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
 
}

void random_ints(int* source, int size)
{
    for (int i=0;i<size;i++)
    {
        source[i]= rand()%100;

    }
}

void show_array(int* source, int size)
{
    for (int i=0;i<size;i++)
    {
        printf("%d\t",source[i]);
    }
}

int main()
{
    srand(time(NULL));
    int* a;
    int* b;
    int* c;

    int* dev_a;
    int* dev_b;
    int* dev_c;

    int size = NUM * sizeof(int);

    hipMalloc((void**)&dev_a, size);
    hipMalloc((void**)&dev_b, size);
    hipMalloc((void**)&dev_c, size);

    a = (int*) malloc(size);
    random_ints(a, NUM);
    b = (int*) malloc(size);
    random_ints(b, NUM);
    c = (int*) malloc(size);

    show_array(a,NUM);

    printf("\n\n");

    show_array(b,NUM);
    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    add <<< NUM,1 >>> (dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);
    

    printf("Result:\n");
    show_array(c,NUM);
    return 0;
}